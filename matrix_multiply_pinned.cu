﻿#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <cstdlib>


int main(int argc, char** argv) {
int N = 1000;
    
    if(argc == 2)
    {
       N = atoi(argv[1]);
    }
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Выделение pinned памяти на CPU
    float* h_matrix;
    hipHostMalloc((void**)&h_matrix, N * N * sizeof(float), hipHostMallocDefault);

    // Генерация случайных данных на CPU
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dis(0.0, 1.0);
    for (int i = 0; i < N * N; ++i) {
        h_matrix[i] = dis(gen);
    }

    // Выделение памяти на GPU
    float* d_matrix;
    hipMalloc((void**)&d_matrix, N * N * sizeof(float));

    // Передача данных с CPU на GPU
    hipMemcpy(d_matrix, h_matrix, N * N * sizeof(float), hipMemcpyHostToDevice);


    // Проверка на ошибки CUDA
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return 1;
    }

    // ... (здесь можно выполнить вычисления на GPU с использованием d_matrix) ...

    // Освобождение памяти на GPU
    hipFree(d_matrix);

    // Освобождение pinned памяти на CPU
    hipHostFree(h_matrix);

    std::cout << "Передача матрицы завершена успешно!" << std::endl;
    return 0;
}



